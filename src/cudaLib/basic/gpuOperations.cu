#include <gpuOperations.cuh>
#include <cudaHelper.cuh>

void gpuSetDevice(int gpuId)
{
    int gpuCount;
    cudaCheck(hipGetDeviceCount(& gpuCount));
    if (gpuCount == 0) {
        fprintf(stderr, "There is no CUDA device.\n");
    }
    if (gpuId >= gpuCount) {
        fprintf(stderr, "GPU id is %d. But there are only %d GPUs.\n", gpuId, gpuCount);
    }

    cudaCheck(hipSetDevice(gpuId));

    printf("Total number of GPUs: %d, Using GPU id: %d\n", gpuCount, gpuId);
}

Solver ** gpuSolverInit(int numMovingFluids, Solver ** solverList, Solver ** ptrContainerList)
{
    Solver ** gpuSolverList = new Solver * [numMovingFluids + 1];

    for (int i = 0; i < numMovingFluids + 1; i++) {
        cudaCheck(hipMalloc(& ptrContainerList[i]->w,           dim * sizeof(float)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->ciX,         dim * sizeof(int)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->ciY,         dim * sizeof(int)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->inversed,    dim * sizeof(int)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->MinvSM,      dim * dim * sizeof(float)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->MinvMrot,    dim * dim * sizeof(float)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->MinvMrotInv, dim * dim * sizeof(float)));

        cudaCheck(hipMemcpy(ptrContainerList[i]->w,           solverList[i]->w,           dim * sizeof(float),       hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->ciX,         solverList[i]->ciX,         dim * sizeof(int),         hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->ciY,         solverList[i]->ciY,         dim * sizeof(int),         hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->inversed,    solverList[i]->inversed,    dim * sizeof(int),         hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->MinvSM,      solverList[i]->MinvSM,      dim * dim * sizeof(float), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->MinvMrot,    solverList[i]->MinvMrot,    dim * dim * sizeof(float), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->MinvMrotInv, solverList[i]->MinvMrotInv, dim * dim * sizeof(float), hipMemcpyHostToDevice));

        cudaCheck(hipMalloc(& gpuSolverList[i], sizeof(Solver)));
        cudaCheck(hipMemcpy(gpuSolverList[i], ptrContainerList[i], sizeof(Solver), hipMemcpyHostToDevice));
    }

    return gpuSolverList;
}

Fluid ** gpuFluidInit(int numMovingFluids, Fluid ** fluidList, Fluid ** ptrContainerList)
{
    Fluid ** gpuFluidList = new Fluid * [numMovingFluids + 1];

    for (int i = 0; i < numMovingFluids + 1; i++) {
        int currNAll = fluidList[i]->nAll;
        // Quantities for LBM
        cudaCheck(hipMalloc(& ptrContainerList[i]->density1,           currNAll * sizeof(float)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->status1,            currNAll * sizeof(Status)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->vel2,           2 * currNAll * sizeof(float)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->force2,         2 * currNAll * sizeof(float)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->fNewDim,      dim * currNAll * sizeof(float)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->fOldDim,      dim * currNAll * sizeof(float)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->gridForceDim, dim * currNAll * sizeof(float)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->equDim,       dim * currNAll * sizeof(float)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->streamedDim,  dim * currNAll * sizeof(bool)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->rotMatrix,                9 * sizeof(float)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->rotMatrixInv,             9 * sizeof(float)));
        cudaCheck(hipMemcpy(ptrContainerList[i]->density1,     fluidList[i]->density1,           currNAll * sizeof(float), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->status1,      fluidList[i]->status1 ,           currNAll * sizeof(Status), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->vel2,         fluidList[i]->vel2 ,          2 * currNAll * sizeof(float), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->force2,       fluidList[i]->force2 ,        2 * currNAll * sizeof(float), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->fNewDim,      fluidList[i]->fNewDim ,     dim * currNAll * sizeof(float), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->fOldDim,      fluidList[i]->fOldDim ,     dim * currNAll * sizeof(float), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->gridForceDim, fluidList[i]->gridForceDim, dim * currNAll * sizeof(float), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->equDim,       fluidList[i]->equDim ,      dim * currNAll * sizeof(float), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->streamedDim,  fluidList[i]->streamedDim,  dim * currNAll * sizeof(bool),  hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->rotMatrix,    fluidList[i]->rotMatrix,                 9 * sizeof(float), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->rotMatrixInv, fluidList[i]->rotMatrixInv,              9 * sizeof(float), hipMemcpyHostToDevice));
        // Three moments for bubble function
        cudaCheck(hipMalloc(& ptrContainerList[i]->kxy, currNAll * sizeof(float)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->kxx, currNAll * sizeof(float)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->kyy, currNAll * sizeof(float)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->pix, currNAll * sizeof(float)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->piy, currNAll * sizeof(float)));
        cudaCheck(hipMemcpy(ptrContainerList[i]->kxy, fluidList[i]->kxy, currNAll * sizeof(float), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->kxx, fluidList[i]->kxx, currNAll * sizeof(float), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->kyy, fluidList[i]->kyy, currNAll * sizeof(float), hipMemcpyHostToDevice));
        // Three moments for 2018 bubble function
        cudaCheck(hipMalloc(& ptrContainerList[i]->c20, currNAll * sizeof(float)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->c02, currNAll * sizeof(float)));
        cudaCheck(hipMalloc(& ptrContainerList[i]->c11, currNAll * sizeof(float)));
        cudaCheck(hipMemcpy(ptrContainerList[i]->c20, fluidList[i]->c20, currNAll * sizeof(float), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->c02, fluidList[i]->c02, currNAll * sizeof(float), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(ptrContainerList[i]->c11, fluidList[i]->c11, currNAll * sizeof(float), hipMemcpyHostToDevice));
        // For overset Grid
        if (i == 0) {
            cudaCheck(hipMalloc(& ptrContainerList[i]->densityOld1,    currNAll * sizeof(float)));
            cudaCheck(hipMalloc(& ptrContainerList[i]->velOld2,    2 * currNAll * sizeof(float)));
            cudaCheck(hipMalloc(& ptrContainerList[i]->fLastDim, dim * currNAll * sizeof(float)));
            cudaCheck(hipMemcpy(ptrContainerList[i]->densityOld1, fluidList[i]->density1,      currNAll * sizeof(float), hipMemcpyHostToDevice));
            cudaCheck(hipMemcpy(ptrContainerList[i]->velOld2,     fluidList[i]->vel2,      2 * currNAll * sizeof(float), hipMemcpyHostToDevice));
            cudaCheck(hipMemcpy(ptrContainerList[i]->fLastDim,    fluidList[i]->fNewDim, dim * currNAll * sizeof(float), hipMemcpyHostToDevice));
        }
        // Combined together
        cudaCheck(hipMalloc(& gpuFluidList[i], sizeof(Fluid)));
        cudaCheck(hipMemcpy(gpuFluidList[i], ptrContainerList[i], sizeof(Fluid), hipMemcpyHostToDevice));
    }

    return gpuFluidList;
}

void gpuSolverFree(int numMovingFluids, Solver ** solverList, Solver ** ptrContainerList, Solver ** gpuSolverList)
{
    for (int i = 0; i < numMovingFluids + 1; i++) {
        cudaCheck(hipFree(ptrContainerList[i]->w));
        cudaCheck(hipFree(ptrContainerList[i]->ciX));
        cudaCheck(hipFree(ptrContainerList[i]->ciY));
        cudaCheck(hipFree(ptrContainerList[i]->inversed));
        cudaCheck(hipFree(ptrContainerList[i]->MinvSM));
        cudaCheck(hipFree(ptrContainerList[i]->MinvMrot));
        cudaCheck(hipFree(ptrContainerList[i]->MinvMrotInv));

        cudaCheck(hipFree(gpuSolverList[i]));
    }
}

void gpuFluidFree(int numMovingFluids, Fluid ** fluidList, Fluid ** ptrContainerList, Fluid ** gpuFluidList)
{
    for (int i = 0; i < numMovingFluids + 1; i++) {
        cudaCheck(hipFree(ptrContainerList[i]->density1));
        cudaCheck(hipFree(ptrContainerList[i]->status1));
        cudaCheck(hipFree(ptrContainerList[i]->vel2));
        cudaCheck(hipFree(ptrContainerList[i]->force2));
        cudaCheck(hipFree(ptrContainerList[i]->fNewDim));
        cudaCheck(hipFree(ptrContainerList[i]->fOldDim));
        cudaCheck(hipFree(ptrContainerList[i]->gridForceDim));
        cudaCheck(hipFree(ptrContainerList[i]->equDim));
        cudaCheck(hipFree(ptrContainerList[i]->streamedDim));
        cudaCheck(hipFree(ptrContainerList[i]->rotMatrix));
        cudaCheck(hipFree(ptrContainerList[i]->rotMatrixInv));

        if (i == 0) {
            cudaCheck(hipFree(ptrContainerList[i]->densityOld1));
            cudaCheck(hipFree(ptrContainerList[i]->velOld2));
            cudaCheck(hipFree(ptrContainerList[i]->fLastDim));
        }

        cudaCheck(hipFree(gpuFluidList[i]));
    }
}

void solverMrotCpyCpu2Gpu(Solver * solver, Solver * ptrContainer)
{
    cudaCheck(hipMemcpy(ptrContainer->MinvMrot,    solver->MinvMrot,    dim * dim * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(ptrContainer->MinvMrotInv, solver->MinvMrotInv, dim * dim * sizeof(float), hipMemcpyHostToDevice));
}

void fluidProperityCpyCpu2Gpu(Fluid * fluid, Fluid * ptrContainer, Fluid * gpuFluid)
{
    cudaCheck(hipMemcpy(ptrContainer->rotMatrix,    fluid->rotMatrix,    9 * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(ptrContainer->rotMatrixInv, fluid->rotMatrixInv, 9 * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(gpuFluid, ptrContainer, sizeof(Fluid), hipMemcpyHostToDevice));
}

void fluidVel2CpyGpu2Cpu(Fluid * fluid, Fluid * ptrContainer)
{
    cudaCheck(hipMemcpy(fluid->vel2, ptrContainer->vel2, 2 * fluid->nAll * sizeof(float), hipMemcpyDeviceToHost));
}

void fluidFDimCpyGpu2Cpu(Fluid * fluid, Fluid * ptrContainer)
{
    cudaCheck(hipMemcpy(fluid->fNewDim, ptrContainer->fNewDim, dim * fluid->nAll * sizeof(float), hipMemcpyDeviceToHost));
}

void fluidStatus1CpyGpu2Cpu(Fluid * fluid, Fluid * ptrContainer)
{
    cudaCheck(hipMemcpy(fluid->status1, ptrContainer->status1, fluid->nAll * sizeof(Status), hipMemcpyDeviceToHost));
}

void fluidStreamedDimCpyGpu2Cpu(Fluid * fluid, Fluid * ptrContainer)
{
    cudaCheck(hipMemcpy(fluid->streamedDim, ptrContainer->streamedDim, dim * fluid->nAll * sizeof(bool), hipMemcpyDeviceToHost));
}

// ---------- For data output ----------
void gpuCopyFluidFOldDim(Fluid * fluid, Fluid * ptrContainer)
{
    cudaCheck(hipMemcpy(fluid->fOldDim, ptrContainer->fOldDim, dim * fluid->nAll * sizeof(float), hipMemcpyDeviceToHost));
}

void gpuCopyFluidGridForceDim(Fluid * fluid, Fluid * ptrContainer)
{
    cudaCheck(hipMemcpy(fluid->gridForceDim, ptrContainer->gridForceDim, dim * fluid->nAll * sizeof(float), hipMemcpyDeviceToHost));
}

void gpuCopyFluidEquDim(Fluid * fluid, Fluid * ptrContainer)
{
    cudaCheck(hipMemcpy(fluid->equDim, ptrContainer->equDim, dim * fluid->nAll * sizeof(float), hipMemcpyDeviceToHost));
}

// ---------- For overset grid ----------
void solverCpy2Gpu(Solver * solver, Solver * ptrContainer)
{
    cudaCheck(hipMemcpy(ptrContainer->ciX, solver->ciX, dim * sizeof(int), hipMemcpyHostToDevice));
}
